#include "hip/hip_runtime.h"
extern "C" {
	#include "lua.h"
	#include "lualib.h"
	#include "lauxlib.h"
}

#include "luaT.h"
#include "THC.h"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_math_constants.h>
#include <math_functions.h>
#include <stdint.h>
#include <unistd.h>

#define TB 256
#define EPS 1e-4

THCState* getCutorchState(lua_State* L)
{
	lua_getglobal(L, "cutorch");
	lua_getfield(L, -1, "getState");
	lua_call(L, 0, 1);
	THCState *state = (THCState*) lua_touserdata(L, -1);
	lua_pop(L, 2);
	return state;
}

void checkCudaError(lua_State *L) {
	hipError_t status = hipPeekAtLastError();
	if (status != hipSuccess) {
		luaL_error(L, hipGetErrorString(status));
	}
}

THCudaTensor *new_tensor_like(THCState *state, THCudaTensor *x)
{
	THCudaTensor *y = THCudaTensor_new(state);
	THCudaTensor_resizeAs(state, y, x);
	return y;
}

__global__ void matting_laplacian_kernel(
	float *input, float *grad, int h, int w, 
	int *CSR_rowIdx, int *CSR_colIdx, float *CSR_val,
	int N
)
{	
	int size = h * w;
	int _id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (_id < size) {
		int x = _id % w, y = _id / w;
		int id = x * h + y;

		/// Because matting laplacian L is systematic, sum row is sufficient
		// 1.1 Binary search
		int start = 0;
		int end = N-1;
		int mid = (start + end)/2;
		int index = -1;
		while (start <= end) {
			int rowIdx = (CSR_rowIdx[mid]) - 1;
			
			if (rowIdx == id) {
				index = mid;    break;
			}
			if (rowIdx > id) {
				end = mid - 1;
				mid = (start + end)/2;
			} else {
				start = mid + 1;
				mid = (start + end)/2;
			}
		}
		if (index != -1) {
			// 1.2 Complete range
			int index_s = index, index_e = index;
			while ( index_s >= 0 && ((CSR_rowIdx[index_s] - 1) == id) )
				index_s--;
			while ( index_e <  N && ((CSR_rowIdx[index_e] - 1) == id) )
				index_e++;
			// 1.3 Sum this row
			for (int i = index_s + 1; i < index_e; i++) {
				//int rowIdx = CSR_rowIdx[i] - 1; 
				int _colIdx = (CSR_colIdx[i]) - 1;
				float val  = CSR_val[i];

				int _x = _colIdx / h, _y = _colIdx % h;
				int colIdx = _y *w + _x;

				grad[_id] 			+= 2*val * input[colIdx];
				grad[_id + size] 	+= 2*val * input[colIdx + size];
				grad[_id + 2*size]  += 2*val * input[colIdx + 2*size];
			}

		}	
	}

	return ;
}
 
//cuda_utils.matting_laplacian(input, h, w, CSR_rowIdx, CSR_colIdx, CSR_val, CSC_rowIdx, CSC_colIdx, CSC_val, N)
int matting_laplacian(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *input 			= (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	int h 							= luaL_checknumber(L, 2);
	int w 							= luaL_checknumber(L, 3);
	THCudaIntTensor *CSR_rowIdx		= (THCudaIntTensor*)luaT_checkudata(L, 4, "torch.CudaIntTensor");
	THCudaIntTensor *CSR_colIdx		= (THCudaIntTensor*)luaT_checkudata(L, 5, "torch.CudaIntTensor");
	THCudaTensor *CSR_val			= (THCudaTensor*)luaT_checkudata(L, 6, "torch.CudaTensor");
	int N 							= luaL_checknumber(L, 7);
	THCudaTensor *grad = new_tensor_like(state, input);
	THCudaTensor_zero(state, grad);

	matting_laplacian_kernel<<<(h*w-1)/TB+1, TB>>>(
		THCudaTensor_data(state, input),
		THCudaTensor_data(state, grad),
		h, w,
		THCudaIntTensor_data(state, CSR_rowIdx),
		THCudaIntTensor_data(state, CSR_colIdx),
		THCudaTensor_data(state, CSR_val),
		N
	);
	checkCudaError(L);
	luaT_pushudata(L, grad, "torch.CudaTensor");
	return 1;
}

__device__ bool InverseMat4x4(double m_in[4][4], double inv_out[4][4]) {
	double m[16], inv[16];
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			m[i * 4 + j] = m_in[i][j];
		}
	}

    inv[0] = m[5]  * m[10] * m[15] - 
             m[5]  * m[11] * m[14] - 
             m[9]  * m[6]  * m[15] + 
             m[9]  * m[7]  * m[14] +
             m[13] * m[6]  * m[11] - 
             m[13] * m[7]  * m[10];

    inv[4] = -m[4]  * m[10] * m[15] + 
              m[4]  * m[11] * m[14] + 
              m[8]  * m[6]  * m[15] - 
              m[8]  * m[7]  * m[14] - 
              m[12] * m[6]  * m[11] + 
              m[12] * m[7]  * m[10];

    inv[8] = m[4]  * m[9] * m[15] - 
             m[4]  * m[11] * m[13] - 
             m[8]  * m[5] * m[15] + 
             m[8]  * m[7] * m[13] + 
             m[12] * m[5] * m[11] - 
             m[12] * m[7] * m[9];

    inv[12] = -m[4]  * m[9] * m[14] + 
               m[4]  * m[10] * m[13] +
               m[8]  * m[5] * m[14] - 
               m[8]  * m[6] * m[13] - 
               m[12] * m[5] * m[10] + 
               m[12] * m[6] * m[9];

    inv[1] = -m[1]  * m[10] * m[15] + 
              m[1]  * m[11] * m[14] + 
              m[9]  * m[2] * m[15] - 
              m[9]  * m[3] * m[14] - 
              m[13] * m[2] * m[11] + 
              m[13] * m[3] * m[10];

    inv[5] = m[0]  * m[10] * m[15] - 
             m[0]  * m[11] * m[14] - 
             m[8]  * m[2] * m[15] + 
             m[8]  * m[3] * m[14] + 
             m[12] * m[2] * m[11] - 
             m[12] * m[3] * m[10];

    inv[9] = -m[0]  * m[9] * m[15] + 
              m[0]  * m[11] * m[13] + 
              m[8]  * m[1] * m[15] - 
              m[8]  * m[3] * m[13] - 
              m[12] * m[1] * m[11] + 
              m[12] * m[3] * m[9];

    inv[13] = m[0]  * m[9] * m[14] - 
              m[0]  * m[10] * m[13] - 
              m[8]  * m[1] * m[14] + 
              m[8]  * m[2] * m[13] + 
              m[12] * m[1] * m[10] - 
              m[12] * m[2] * m[9];

    inv[2] = m[1]  * m[6] * m[15] - 
             m[1]  * m[7] * m[14] - 
             m[5]  * m[2] * m[15] + 
             m[5]  * m[3] * m[14] + 
             m[13] * m[2] * m[7] - 
             m[13] * m[3] * m[6];

    inv[6] = -m[0]  * m[6] * m[15] + 
              m[0]  * m[7] * m[14] + 
              m[4]  * m[2] * m[15] - 
              m[4]  * m[3] * m[14] - 
              m[12] * m[2] * m[7] + 
              m[12] * m[3] * m[6];

    inv[10] = m[0]  * m[5] * m[15] - 
              m[0]  * m[7] * m[13] - 
              m[4]  * m[1] * m[15] + 
              m[4]  * m[3] * m[13] + 
              m[12] * m[1] * m[7] - 
              m[12] * m[3] * m[5];

    inv[14] = -m[0]  * m[5] * m[14] + 
               m[0]  * m[6] * m[13] + 
               m[4]  * m[1] * m[14] - 
               m[4]  * m[2] * m[13] - 
               m[12] * m[1] * m[6] + 
               m[12] * m[2] * m[5];

    inv[3] = -m[1] * m[6] * m[11] + 
              m[1] * m[7] * m[10] + 
              m[5] * m[2] * m[11] - 
              m[5] * m[3] * m[10] - 
              m[9] * m[2] * m[7] + 
              m[9] * m[3] * m[6];

    inv[7] = m[0] * m[6] * m[11] - 
             m[0] * m[7] * m[10] - 
             m[4] * m[2] * m[11] + 
             m[4] * m[3] * m[10] + 
             m[8] * m[2] * m[7] - 
             m[8] * m[3] * m[6];

    inv[11] = -m[0] * m[5] * m[11] + 
               m[0] * m[7] * m[9] + 
               m[4] * m[1] * m[11] - 
               m[4] * m[3] * m[9] - 
               m[8] * m[1] * m[7] + 
               m[8] * m[3] * m[5];

    inv[15] = m[0] * m[5] * m[10] - 
              m[0] * m[6] * m[9] - 
              m[4] * m[1] * m[10] + 
              m[4] * m[2] * m[9] + 
              m[8] * m[1] * m[6] - 
              m[8] * m[2] * m[5];

    double det = m[0] * inv[0] + m[1] * inv[4] + m[2] * inv[8] + m[3] * inv[12];

    if (abs(det) < 1e-9) {
        return false;
    }


    det = 1.0 / det; 

    for (int i = 0; i < 4; i++) {
    	for (int j = 0; j < 4; j++) {
    		inv_out[i][j] = inv[i * 4 + j] * det;
    	}
    }

    return true;
}

__global__ void best_local_affine_kernel(
	float *output, float *input, float *affine_model,
	int h, int w, float epsilon, int kernel_radius
)
{
	int size = h * w;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (id < size) {
		int x = id % w, y = id / w;
		
		double Mt_M[4][4] = {}; // 4x4
		double invMt_M[4][4] = {};
		double Mt_S[3][4] = {}; // RGB -> 1x4
		double A[3][4] = {}; 
		for (int i = 0; i < 4; i++) 
			for (int j = 0; j < 4; j++) {
				Mt_M[i][j] = 0, invMt_M[i][j] = 0;
				if (i != 3) {
					Mt_S[i][j] = 0, A[i][j] = 0;
					if (i == j)
			    		Mt_M[i][j] = 1e-3;
			    }
			}

		for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
			for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
				
				int xx = x + dx, yy = y + dy;
				int id2 = yy * w + xx;

				if (0 <= xx && xx < w && 0 <= yy && yy < h) {

					Mt_M[0][0] += input[id2 + 2*size] * input[id2 + 2*size];
					Mt_M[0][1] += input[id2 + 2*size] * input[id2 + size];
					Mt_M[0][2] += input[id2 + 2*size] * input[id2];
					Mt_M[0][3] += input[id2 + 2*size];

					Mt_M[1][0] += input[id2 + size] * input[id2 + 2*size];
					Mt_M[1][1] += input[id2 + size] * input[id2 + size];
					Mt_M[1][2] += input[id2 + size] * input[id2];
					Mt_M[1][3] += input[id2 + size];

					Mt_M[2][0] += input[id2] * input[id2 + 2*size];
					Mt_M[2][1] += input[id2] * input[id2 + size];
					Mt_M[2][2] += input[id2] * input[id2];
					Mt_M[2][3] += input[id2];
					 
					Mt_M[3][0] += input[id2 + 2*size];
					Mt_M[3][1] += input[id2 + size];
					Mt_M[3][2] += input[id2];
					Mt_M[3][3] += 1;

					Mt_S[0][0] += input[id2 + 2*size] * output[id2 + 2*size];
					Mt_S[0][1] += input[id2 + size] * output[id2 + 2*size];
					Mt_S[0][2] += input[id2] * output[id2 + 2*size];
					Mt_S[0][3] += output[id2 + 2*size];

					Mt_S[1][0] += input[id2 + 2*size] * output[id2 + size];
					Mt_S[1][1] += input[id2 + size] * output[id2 + size];
					Mt_S[1][2] += input[id2] * output[id2 + size];
					Mt_S[1][3] += output[id2 + size];

					Mt_S[2][0] += input[id2 + 2*size] * output[id2];
					Mt_S[2][1] += input[id2 + size] * output[id2];
					Mt_S[2][2] += input[id2] * output[id2];
					Mt_S[2][3] += output[id2];
				}
			}
		}

		bool success = InverseMat4x4(Mt_M, invMt_M);

		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 4; j++) {
				for (int k = 0; k < 4; k++) {
					A[i][j] += invMt_M[j][k] * Mt_S[i][k];
				}
			}
		}

		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 4; j++) {
				int affine_id = i * 4 + j;
				affine_model[12 * id + affine_id] = A[i][j];
			}
		}



	}
	return ;
}


__global__ void bilateral_smooth_kernel(
	float *affine_model, float *filtered_affine_model, float *guide, 
	int h, int w, int kernel_radius, float sigma1, float sigma2
)	
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w;
	if (id < size) {
		int x = id % w;
		int y = id / w;

		double sum_affine[12] = {};
		double sum_weight = 0;
		for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
			for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
				int yy = y + dy, xx = x + dx;
				int id2 = yy * w + xx;
				if (0 <= xx && xx < w && 0 <= yy && yy < h) {
					float color_diff1 = guide[yy*w + xx] - guide[y*w + x];
					float color_diff2 = guide[yy*w + xx + size] - guide[y*w + x + size];
					float color_diff3 = guide[yy*w + xx + 2*size] - guide[y*w + x + 2*size];
					float color_diff_sqr = 
						(color_diff1*color_diff1 + color_diff2*color_diff2 + color_diff3*color_diff3) / 3;

					float v1 = exp(-(dx * dx + dy * dy) / (2 * sigma1 * sigma1));
					float v2 = exp(-(color_diff_sqr) / (2 * sigma2 * sigma2));
					float weight = v1 * v2;

					for (int i = 0; i < 3; i++) {
						for (int j = 0; j < 4; j++) {
							int affine_id = i * 4 + j;
							sum_affine[affine_id] += weight * affine_model[id2*12 + affine_id];
						}
					}
					sum_weight += weight;
				}
			}
		}

		for (int i = 0; i < 3; i++) {
			for (int j = 0; j < 4; j++) {
				int affine_id = i * 4 + j;
				filtered_affine_model[id*12 + affine_id] = sum_affine[affine_id] / sum_weight;
			}
		}
	}
	return ;
}


__global__ void reconstruction_best_kernel(
	float *input, float *filtered_affine_model, float *filtered_best_output,
	int h, int w
) 
{	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w;
	if (id < size) {
		double out1 = 
			input[id + 2*size] * filtered_affine_model[id*12 + 0] + // A[0][0] + 
			input[id + size]   * filtered_affine_model[id*12 + 1] + // A[0][1] + 
			input[id]          * filtered_affine_model[id*12 + 2] + // A[0][2] +
								 filtered_affine_model[id*12 + 3]; //A[0][3];
		double out2 = 
			input[id + 2*size] * filtered_affine_model[id*12 + 4] + //A[1][0] + 
			input[id + size]   * filtered_affine_model[id*12 + 5] + //A[1][1] + 
			input[id]          * filtered_affine_model[id*12 + 6] + //A[1][2] +
								 filtered_affine_model[id*12 + 7]; //A[1][3];
		double out3 = 
			input[id + 2*size] * filtered_affine_model[id*12 + 8] + //A[2][0] + 
			input[id + size]   * filtered_affine_model[id*12 + 9] + //A[2][1] + 
			input[id]          * filtered_affine_model[id*12 + 10] + //A[2][2] +
								 filtered_affine_model[id*12 + 11]; // A[2][3];

		filtered_best_output[id] = out1;
		filtered_best_output[id + size] = out2;
		filtered_best_output[id + 2*size] = out3;
	}
	return ;
}

// local best01 = cuda_utils.smooth_local_affine(output01, input01, epsilon, patch, h, w, filter_radius, sigma1, sigma2)
int smooth_local_affine(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *output  = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *input   = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	float         epsilon = luaL_checknumber(L, 3);
	int 		  patch   = luaL_checknumber(L, 4);
	int 		  h       = luaL_checknumber(L, 5);
	int 		  w       = luaL_checknumber(L, 6);
	int 		  f_r     = luaL_checknumber(L, 7);
	float 		  sigma1  = luaL_checknumber(L, 8);
	float 		  sigma2  = luaL_checknumber(L, 9);

	THCudaTensor *filtered_best_output = new_tensor_like(state, input);
	THCudaTensor_zero(state, filtered_best_output);

	THCudaTensor *affine_model = THCudaTensor_new(state);
	THCudaTensor_resize2d(state, affine_model, h*w, 12);
	THCudaTensor_zero(state, affine_model);

	THCudaTensor *filtered_affine_model = THCudaTensor_new(state);
	THCudaTensor_resize2d(state, filtered_affine_model, h*w, 12);
	THCudaTensor_zero(state, filtered_affine_model);

	int radius = (patch-1) / 2;

	best_local_affine_kernel<<<(h*w)/TB+1, TB>>>(
		THCudaTensor_data(state, output),
		THCudaTensor_data(state, input),
		THCudaTensor_data(state, affine_model),
		h, w, epsilon, radius
	);
	checkCudaError(L);

	bilateral_smooth_kernel<<<(h*w)/TB+1, TB>>>(
		THCudaTensor_data(state, affine_model),
		THCudaTensor_data(state, filtered_affine_model),
		THCudaTensor_data(state, input),
		h, w, f_r, sigma1, sigma2
	);
	checkCudaError(L);
	THCudaTensor_free(state, affine_model);

	reconstruction_best_kernel<<<(h*w)/TB+1, TB>>>(
		THCudaTensor_data(state, input),
		THCudaTensor_data(state, filtered_affine_model),
		THCudaTensor_data(state, filtered_best_output),
		h, w
	);
	checkCudaError(L);
	THCudaTensor_free(state, filtered_affine_model);
	

	luaT_pushudata(L, filtered_best_output, "torch.CudaTensor");

	return 1;
}

static const struct luaL_Reg funcs[] = {
	{"matting_laplacian", matting_laplacian},
	{"smooth_local_affine", smooth_local_affine},
	{NULL, NULL}
};

extern "C" int luaopen_libcuda_utils(lua_State *L) {
	luaL_openlib(L, "cuda_utils", funcs, 0);
	return 1;
}